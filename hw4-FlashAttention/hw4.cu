
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <sys/time.h>
#define BLOCK_SIZE 32

void input(char *input_filename);
void output(char *output_filename);
void flash_attention(float *q, float *k, float *v, float *o);

__global__ void flash_attention_kenel(float *q, float *k, float *m, float *l, float *o, float *v, int d, float scalar, int grid_size);
__device__ void QKDotAndScalar(float *out, float *q, float *k, int d, float scalar);
__device__ void RowMax(float *out, float *in);
__device__ void MinusMaxAndExp(float *out, float *in, float *mx );
__device__ void RowSum(float *out, float *in);
__device__ void UpdateMiLiOi(float *shared_m, float *shared_l, float *o, float *mij, float *lij, float *pij, float *shared_v , int d);

float _max(float a, float b) { return a > b ? a : b; }
float _min(float a, float b) { return a < b ? a : b; }
double getTimeStamp() {
    struct timeval tv;
    gettimeofday( &tv, NULL );
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

int B, N, d;
float *Q, *K, *V, *O;
float *device_q, *device_k, *device_v, *device_o;
float *device_l, *device_m;
int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <input_filename> <output_filename>\n", argv[0]);
        return 1;
    }

    input(argv[1]);

    double start, end;
    start = getTimeStamp();

    hipMalloc(&device_q, N * d * sizeof(float));
    hipMalloc(&device_k, N * d * sizeof(float));
    hipMalloc(&device_v, N * d * sizeof(float));
    hipMalloc(&device_o, N * d * sizeof(float));
    hipMalloc(&device_l, N * sizeof(float));
    hipMalloc(&device_m, N * sizeof(float));

    for (int i = 0; i < B; i++) {
        flash_attention(
            Q + (i * N * d), 
            K + (i * N * d), 
            V + (i * N * d), 
            O + (i * N * d)
        );
    }

    end = getTimeStamp();
    printf("(B, N, d): (%d, %d, %d)\n", B, N, d);
    printf("Time: %.3f seconds\n", end - start);

    output(argv[2]);
    // cudaFree(device_q);
    // cudaFree(device_k);
    // cudaFree(device_v);
    // cudaFree(device_o);
    // cudaFree(device_l);
    // cudaFree(device_m);
    return 0;
}

void input(char *input_filename) {
    FILE *file = fopen(input_filename, "rb");

    fread(&B, sizeof(int), 1, file);
    fread(&N, sizeof(int), 1, file);
    fread(&d, sizeof(int), 1, file);

    Q = (float *)malloc(B * N * d * sizeof(float));
    K = (float *)malloc(B * N * d * sizeof(float));
    V = (float *)malloc(B * N * d * sizeof(float));
    O = (float *)malloc(B * N * d * sizeof(float));

    for (int i = 0; i < B; i++) {
        fread(Q + (i * N * d), sizeof(float), N * d, file);
        fread(K + (i * N * d), sizeof(float), N * d, file);
        fread(V + (i * N * d), sizeof(float), N * d, file);
    }
    memset(O, 0x00, B * N * d * sizeof(float));

    fclose(file);
}

void output(char *output_filename) {
    FILE *file = fopen(output_filename, "wb");

    fwrite(O, sizeof(float), B * N * d, file);

    free(Q);
    free(K);
    free(V);
    free(O);

    fclose(file);
}

void flash_attention(float *q, float *k, float *v, float *o) {
    // float *device_q, *device_k, *device_v, *device_o;
    // float *device_l, *device_m;
    float *l = (float *)malloc(N * sizeof(float));
    float *m = (float *)malloc(N * sizeof(float));

    memset(l, 0x00, N * sizeof(float));
    for (int i = 0; i < N; i++) {
        m[i] = FLT_MIN;
    }

    // cudaMalloc(&device_q, N * d * sizeof(float));
    // cudaMalloc(&device_k, N * d * sizeof(float));
    // cudaMalloc(&device_v, N * d * sizeof(float));
    // cudaMalloc(&device_o, N * d * sizeof(float));
    // cudaMalloc(&device_l, N * sizeof(float));
    // cudaMalloc(&device_m, N * sizeof(float));


    hipMemcpy(device_q, q, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_k, k, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_v, v, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_o, o, N * d * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_l, l, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_m, m, N * sizeof(float), hipMemcpyHostToDevice);

    int grid_size = N / BLOCK_SIZE;
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(grid_size);
   
    flash_attention_kenel<<< grid, block >>>(device_q, device_k, device_m, device_l , device_o, device_v, d , 1.0 / sqrt(d) , grid_size);
    
    hipMemcpy(o, device_o, N * d * sizeof(float), hipMemcpyDeviceToHost);
}
__global__ void flash_attention_kenel(float *q, float *k, float *m, float *l, float *o, float *v, int d, float scalar , int grid_size){
    __shared__ float shared_q[BLOCK_SIZE * 64];
    __shared__ float shared_k[BLOCK_SIZE * 64];
    __shared__ float shared_v[BLOCK_SIZE * 64];
    __shared__ float shared_m[BLOCK_SIZE];
    __shared__ float shared_l[BLOCK_SIZE];
    __shared__ float shared_o[BLOCK_SIZE * 64]; 
    __shared__ float sij[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ float mij[BLOCK_SIZE];
    __shared__ float pij[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ float lij[BLOCK_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int start = blockIdx.x;
    int step = d / BLOCK_SIZE;

    for(int i = 0 ; i < step; i++){
        shared_q[ty * d + (i * BLOCK_SIZE + tx)] = q[(start * BLOCK_SIZE + ty) * d + (i * BLOCK_SIZE + tx)];
        shared_o[ty * d + (i * BLOCK_SIZE + tx)] = o[(start * BLOCK_SIZE + ty) * d + (i * BLOCK_SIZE + tx)];
        // shared_k[tx * d + (i * BLOCK_SIZE + ty)] = k[tx * d + (i * BLOCK_SIZE + ty) + j * BLOCK_SIZE * d];
        // shared_v[tx * d + (i * BLOCK_SIZE + ty)] = v[tx * d + (i * BLOCK_SIZE + ty) + j * BLOCK_SIZE * d];
    }
    shared_m[ty] = m[ty + start * BLOCK_SIZE];
    shared_l[ty] = l[ty + start * BLOCK_SIZE];

    for (int j = 0; j < grid_size; j++) {

        for(int i = 0 ; i < step; i++){
            shared_k[tx * d + (i * BLOCK_SIZE + ty)] = k[tx * d + (i * BLOCK_SIZE + ty) + j * BLOCK_SIZE * d];
            shared_v[tx * d + (i * BLOCK_SIZE + ty)] = v[tx * d + (i * BLOCK_SIZE + ty) + j * BLOCK_SIZE * d];
        }
         __syncthreads();

        QKDotAndScalar(sij , shared_q , shared_k , d , scalar);

        __syncthreads();

        RowMax(mij , sij);

        __syncthreads();

        MinusMaxAndExp(pij , sij , mij);

        __syncthreads();

        RowSum(lij , pij);

        __syncthreads();
        
        UpdateMiLiOi(shared_m, shared_l, shared_o, mij, lij, pij, shared_v, d);
    }
    for(int i = 0 ; i < step; i++){
        o[(start * BLOCK_SIZE + ty) * d + (i * BLOCK_SIZE + tx)] =  shared_o[ty * d + (i * BLOCK_SIZE + tx)];
    }
    // m[ty + start * BLOCK_SIZE] = shared_m[ty];
    // l[ty + start * BLOCK_SIZE] = shared_l[ty];
}
__device__ void QKDotAndScalar(float *out, float *q, float *k, int d, float scalar) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float result = 0.0F;

    for (int t = 0; t < d ; t++) {
        result += q[ty * d + t] * k[tx * d + t];
    }
    result *= scalar;
    out[ty * BLOCK_SIZE + tx] = result;
}

__device__ void RowMax(float *out, float *in) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    if(tx == 0){
        float result = in[ty  * BLOCK_SIZE];
        for (int j = 0; j < BLOCK_SIZE; j++) {
            result = max(result, in[ty * BLOCK_SIZE + j]);
        }
        out[ty] = result;
    }
}

__device__ void MinusMaxAndExp(float *out, float *in, float *mx ) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    out[ty  * BLOCK_SIZE + tx] = exp(in[ty * BLOCK_SIZE + tx] - mx[ty]);
}
__device__ void RowSum(float *out, float *in) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    if(tx == 0){
        float result = 0.0F;
        for (int j = 0; j < BLOCK_SIZE; j++) {
            result += in[ty * BLOCK_SIZE + j];
        }
        out[ty] = result;
    }
}

__device__ void UpdateMiLiOi(float *shared_m, float *shared_l, float *shared_o, float *mij, float *lij, float *pij, float *shared_v , int d) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
   
    __shared__ float new_m[BLOCK_SIZE];
    __shared__ float new_l[BLOCK_SIZE];

    float thread_l = shared_l[ty];
    float thread_m = shared_m[ty];
    float thread_mij = mij[ty];

    if(tx == 0){
        new_m[ty] = max(thread_m , thread_mij);
        new_l[ty] = exp(thread_m - new_m[ty]) * thread_l + exp(thread_mij - new_m[ty]) * lij[ty];
    }

    __syncthreads();

    for (int j = 0; j < d / BLOCK_SIZE; j++) {
        float pv = 0.0F;
        for (int t = 0; t < BLOCK_SIZE; t++) {
            pv += pij[ty * BLOCK_SIZE + t] * shared_v[t * d + (j * BLOCK_SIZE + tx)];
        }
        shared_o[ty * d + (j * BLOCK_SIZE + tx)] = (thread_l * exp(thread_m - new_m[ty]) * shared_o[ty * d + (j * BLOCK_SIZE + tx)] + exp(thread_mij - new_m[ty]) * pv) / new_l[ty];
    }

    if(tx == 0){
        shared_m[ty] = new_m[ty];
        shared_l[ty] = new_l[ty];
    }

}
